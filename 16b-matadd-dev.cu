#include "hip/hip_runtime.h"
#include "./common.cpp"
#include <thread>
using std::thread;

// input parameters
unsigned nrow = 10000; // num rows
unsigned ncol = 10000; // num columns

// host-side data
float* matA = nullptr;
float* matB = nullptr;
float* matC = nullptr;

// device-side data
float* dev_matA = nullptr;
float* dev_matB = nullptr;
float* dev_matC = nullptr;

float** arrA = new float*[3];
float** arrB = new float*[3];
float** arrC = new float*[3];

int index_0 = 0;
int index_1 = 1;
int index_2 = 2;

// CUDA kernel function
__global__ void kernel_matadd( float* c, const float* a, const float* b, unsigned nrow, unsigned ncol ) {
	unsigned col = blockIdx.x * blockDim.x + threadIdx.x; // CUDA-provided index
	unsigned row = blockIdx.y * blockDim.y + threadIdx.y; // CUDA-provided index
	if (row < nrow && col < ncol) {
		unsigned i = row * ncol + col; // converted to 1D index
		c[i] = a[i] + b[i];
	}
}

void *cpuFunction1() 
{
	try {
		matA = new float[nrow * ncol];
		matB = new float[nrow * ncol];
		matC = new float[nrow * ncol];

        arrA[index_0] = matA;
        arrB[index_0] = matB;
        arrC[index_0] = matC;

	} catch (const exception& e) {
		printf("C++ EXCEPTION: %s\n", e.what());
		exit(EXIT_FAILURE);
	}
	// set random data
	srand( 0 );
	setNormalizedRandomData( arrA[index_0], nrow * ncol );
	setNormalizedRandomData( arrB[index_0], nrow * ncol );

	return NULL;
}

void *cudaFunction() 
{
	// allocate device memory
	ELAPSED_TIME_BEGIN(1);
	printf("\n\n=====\n\n");
	hipMalloc( (void**)&dev_matA, nrow * ncol * sizeof(float) );
	printf("\n\n=====\n\n");
	hipMalloc( (void**)&dev_matB, nrow * ncol * sizeof(float) );
	hipMalloc( (void**)&dev_matC, nrow * ncol * sizeof(float) );
	CUDA_CHECK_ERROR();
	// copy to device from host
	hipMemcpy( dev_matA, arrA[index_1], nrow * ncol * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dev_matB, arrB[index_1], nrow * ncol * sizeof(float), hipMemcpyHostToDevice );
	CUDA_CHECK_ERROR();
	// CUDA kernel launch
	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid((ncol + dimBlock.x - 1) / dimBlock.x, (nrow + dimBlock.y - 1) / dimBlock.y, 1);
	CUDA_PRINT_CONFIG_2D( ncol, nrow );
	ELAPSED_TIME_BEGIN(0);
	kernel_matadd <<< dimGrid, dimBlock>>>( dev_matC, dev_matA, dev_matB, nrow, ncol );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	CUDA_CHECK_ERROR();
	// copy to host from device
	hipMemcpy( arrC[index_1], dev_matC, nrow * ncol * sizeof(float), hipMemcpyDeviceToHost );
	CUDA_CHECK_ERROR();
	ELAPSED_TIME_END(1);
	// free device memory
	hipFree( dev_matA );
	hipFree( dev_matB );
	hipFree( dev_matC );
	CUDA_CHECK_ERROR();
	
	return NULL;
}

void *cpuFunction2() 
{

	// check the result
	float sumA = getSum( arrA[index_2], nrow * ncol );
	float sumB = getSum( arrB[index_2], nrow * ncol );
	float sumC = getSum( arrC[index_2], nrow * ncol );
	float diff = fabsf( sumC - (sumA + sumB) );
	printf("matrix size = nrow * ncol = %d * %d\n", nrow, ncol);
	printf("sumC = %f\n", sumC);
	printf("sumA = %f\n", sumA);
	printf("sumB = %f\n", sumB);
	printf("diff(sumC, sumA+sumB) =  %f\n", diff);
	printf("diff(sumC, sumA+sumB) / (nrow * ncol) =  %f\n", diff / (nrow * ncol));
	printf("arrC[%d] : ", index_2);
	printMat( "arrC", arrC[index_2], nrow, ncol );
	printf("arrA[%d] : ", index_2);
	printMat( "matA", arrA[index_2], nrow, ncol );
	printf("arrB[%d] : ", index_2);
	printMat( "matB", arrB[index_2], nrow, ncol );
	// cleaning
	delete[] arrA[index_2];
	delete[] arrB[index_2];
	delete[] arrC[index_2];

	return NULL;
}


int main(const int argc, const char* argv[]) {

	index_0 = 0;
    cpuFunction1();
	
	index_0 = 1;
	index_1 = 0;
    cpuFunction1();
    cudaFunction();
    
	index_0 = 2;
	index_1 = 1;
	index_2 = 0;
    cpuFunction1();
    cudaFunction();
	cpuFunction2();
	
	int i = 0;
	
	while(1) 
	{
		printf("\n ----start Cycle---- \n");

        index_0 = (i) % 3;
        index_1 = (i + 1) % 3;
        index_2 = (i + 2) % 3;

		thread t1(cpuFunction1);
		thread t2(cudaFunction);
		thread t3(cpuFunction2);

		t1.join();
		t2.join();
		t3.join();

		i++;
	}

	// done
	return 0;
}

